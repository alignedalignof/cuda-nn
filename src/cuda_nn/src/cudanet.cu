#include "hip/hip_runtime.h"
#include "kernelmat.h"
#include "hiprand/hiprand_kernel.h"

#define K 0.1f

extern "C" __global__ void Cuda2dMemRandNormal(KernelMat* A, float k)
{
	hiprandState_t state;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	hiprand_init(clock(), row, col, &state);
	if (row < A->rows && col < A->cols)
		A->data[row*A->stride + col] = k*hiprand_normal(&state);
}

extern "C" __global__ void Cuda2dMemDiag(KernelMat* A, float k)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (row < A->rows && col < A->cols)
		A->data[row*A->stride + col] = (row == col) ? k : 0;
}

extern "C" __global__ void Cuda2dMemMaxRow(const KernelMat* A, KernelMat* M)
{
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if (col >= A->cols)
		return;
			
	int k = A->rows;
	int stride = A->stride;
	int i = 0;
	float m = A->data[col];
	while (k --> 1)
	{
		float a = A->data[k*stride + col];
		if (a > m)
		{
			m = a;
			i = k;
		}
	}
	M->data[col] = i;
}

__device__ static inline float MatMulCell(const KernelMat* A, const KernelMat* B, int row, int col)
{	
	int n = A->cols;
	const float* a_data = A->data + row*A->stride;
	const float* b_data = B->data + col;
	int b_stride = B->stride;
	
	float acc = 0;
	for (int i = 0; i < n; i++)
		acc += a_data[i]*b_data[i*b_stride];
	return acc;
}

__device__ static inline void MatMadLayer(const KernelMat* A, const KernelMat* B, const KernelMat* C, float a[8][4][32], float4 b[8][32], float4* acc)
{
	int ax = threadIdx.x & 1;
	int ay = threadIdx.x >> 1;
	const float* a_data = A->data + (128*blockIdx.y + 16*threadIdx.y + ay)*A->stride + 4*ax;
	const float* b_data = B->data + threadIdx.y*B->stride + blockIdx.x*128 + 4*threadIdx.x;
	const float* b_end = B->data + B->rows*B->stride;
	const float* a_end = A->data + A->rows*A->stride;
	
	for (int i = 16; i --> 0;)
		acc[i] = float4{ 0, 0, 0, 0 };

	int N = A->cols;
	int b_stride = B->stride;
	while (N > 0)
	{	
		N -= 8;
		
		b[threadIdx.y][threadIdx.x] = (b_data < b_end) ? ((float4*)b_data)[0] : float4{0, 0, 0, 0};
		b_data += 8*b_stride;

		float4 a4 = (a_data < a_end) ? ((float4*)a_data)[0] : float4{ 0, 0, 0, 0 };
		int x = threadIdx.x;
		x = ((x & 1) << 4) | (x >> 1);
		a[threadIdx.y][0][x] = a4.x;
		a[threadIdx.y][1][x] = a4.y;
		a[threadIdx.y][2][x] = a4.z;
		a[threadIdx.y][3][x] = a4.w;
		a_data += 8;
		
		__syncthreads();
		
		#pragma unroll
		for (x = 0; x < 8; x++)
		{
			float4 bx = ((float4*)&b[x])[threadIdx.x];
			#pragma unroll
			for (int y = 0; y < 16; y += 4)
			{
				a4 = ((float4*)&a[threadIdx.y][x & 3][y + ((x & 4) << 2)])[0];

				acc[y].x += a4.x*bx.x;
				acc[y].y += a4.x*bx.y;
				acc[y].z += a4.x*bx.z;
				acc[y].w += a4.x*bx.w;
				
				acc[y + 1].x += a4.y*bx.x;
				acc[y + 1].y += a4.y*bx.y;
				acc[y + 1].z += a4.y*bx.z;
				acc[y + 1].w += a4.y*bx.w;
				
				acc[y + 2].x += a4.z*bx.x;
				acc[y + 2].y += a4.z*bx.y;
				acc[y + 2].z += a4.z*bx.z;
				acc[y + 2].w += a4.z*bx.w;
				
				acc[y + 3].x += a4.w*bx.x;
				acc[y + 3].y += a4.w*bx.y;
				acc[y + 3].z += a4.w*bx.z;
				acc[y + 3].w += a4.w*bx.w;
			}
		}
		__syncthreads();
	}
	
	int row = 128*blockIdx.y + 16*threadIdx.y + threadIdx.x;
	a[threadIdx.y][0][threadIdx.x] = ((row < C->rows) && (threadIdx.x < 16)) ? C->data[row] : 0;
	__syncthreads();
	for (int y = 0; y < 16; y++)
	{
		float c = a[threadIdx.y][0][y];
		acc[y].x += c;
		acc[y].y += c;
		acc[y].z += c;
		acc[y].w += c;
	}
}

extern "C" __global__ void CudaNetReluFwd(const KernelMat* in, const KernelMat* W, const KernelMat* B, KernelMat* out)
{
	__shared__ float4 b[8][32];
	__shared__ __align__(alignof(float4)) float a[8][4][32];
	
	float4 acc[16];
	MatMadLayer(W, in, B, a, b, acc);
	
	int stride = out->stride;
	if (128*blockIdx.x + 4*threadIdx.x >= stride)
		return;
	float* o = out->data + (128*blockIdx.y + 16*threadIdx.y)*stride + 128*blockIdx.x + 4*threadIdx.x;
	float* o_end = out->data + out->rows*stride;
	for (int y = 0; y < 16; y++)
	{
		if (o >= o_end)
			return;
		acc[y].x = (acc[y].x > 0) ? acc[y].x : 0;
		acc[y].y = (acc[y].y > 0) ? acc[y].y : 0;
		acc[y].z = (acc[y].z > 0) ? acc[y].z : 0;
		acc[y].w = (acc[y].w > 0) ? acc[y].w : 0;
		((float4*)o)[0] = acc[y];
		o += stride;
	}
}

extern "C" __global__ void CudaNetSigmoidFwd(const KernelMat* in, const KernelMat* W, const KernelMat* B, KernelMat* out)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	if (row >= out->rows || col >= out->cols)
		return;
		
	float cell = MatMulCell(W, in, row, col);
	cell += B->data[row];
	
	out->data[row*out->stride + col] = 1.0/(1.0 + expf(-cell));
}

extern "C" __global__ void CudaNetSigmoidBwdZ(const KernelMat* A, const KernelMat* Y, KernelMat* dZ)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (row < dZ->rows && col < dZ->cols)
		dZ->data[row*dZ->stride + col] = A->data[row*A->stride + col] - Y->data[row*Y->stride + col];
}

//dZout ~~ W.T*dZin
extern "C" __global__ void CudaNetReluBwdZ(const KernelMat* dZin, const KernelMat* W, KernelMat* A, KernelMat* dZout)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (row >= dZout->rows || col >= dZout->cols)
		return;
		
	int k = dZin->rows;
	const float* w_data = W->data + row;
	const float* dzin_data = dZin->data + col;
	int w_stride = W->stride;
	int dzin_stride = dZin->stride;
		
	float acc = 0;
	for (int i = 0; i < k; i++)
		acc += w_data[i*w_stride]*dzin_data[i*dzin_stride];
	
	dZout->data[row*dZout->stride + col] = (A->data[row*A->stride + col] > 0) ? acc : 0;
}

//B = B - k/m * sum(dZ, axis = 1)
extern "C" __global__ void CudaNetReluBwdB(const KernelMat* dZ, KernelMat* B)
{
	const int m = dZ->cols;
	int row = blockIdx.y;
	int col = threadIdx.x;

	if (row >= dZ->rows)
		return;
	
	const float* dz = dZ->data + row*dZ->stride;
	float acc = 0;
	for (; col < m;  col += blockDim.x)
		acc += dz[col];
	
	__shared__ float accs[32];
	accs[threadIdx.x] = acc;
	__syncwarp();
	
	if (threadIdx.x != 0)
		return;
	acc = 0;
	for (int i = 0; i < 32; i++)
		acc += accs[i];
	B->data[row] -= K*acc/m;
}

//W = W - (k/m)*dZ*A.T
//TODO optimize
extern "C" __global__ void CudaNetReluBwdW(const KernelMat* dZ, const KernelMat* A, KernelMat* W)
{
	int ax = threadIdx.x & 1;
	int ay = threadIdx.x >> 1;
	const float* dz = dZ->data + (128*blockIdx.y + 16*threadIdx.y + ay)*dZ->stride + 4*ax;
	const float* a = A->data + (128*blockIdx.x + 16*threadIdx.y + ay)*A->stride + 4*ax;
	const float* dz_end = dZ->data + dZ->rows*dZ->stride;
	const float* a_end = A->data + A->rows*A->stride;
	
	__shared__ __align__(alignof(float4)) float dz_blk[8][128];
	__shared__ __align__(alignof(float4)) float a_blk[8][128];
	
	float4 acc[16];
	for (int i = 16; i --> 0;)
		acc[i] = float4{ 0, 0, 0, 0 };

	int N = dZ->cols;
	int layer = 4*ax;
	int bank = (16*(threadIdx.y + ax) + ay) & 0x7f;
	while (N > 0)
	{	
		float4 a4 = (a < a_end) ? ((float4*)a)[0] : float4{ 0, 0, 0, 0 };
		a_blk[layer][bank] = a4.x;
		a_blk[layer + 1][bank] = a4.y;
		a_blk[layer + 2][bank] = a4.z;
		a_blk[layer + 3][bank] = a4.w;
		a += 8;

		float4 dz4 = (dz < dz_end) ? ((float4*)dz)[0] : float4{ 0, 0, 0, 0 };
		dz_blk[layer][bank] = dz4.x;
		dz_blk[layer + 1][bank] = dz4.y;
		dz_blk[layer + 2][bank] = dz4.z;
		dz_blk[layer + 3][bank] = dz4.w;
		dz += 8;
		
		__syncthreads();
		
		#pragma unroll
		for (int l = 0; l < 8; l++)
		{
			int ofs = 4*(l & 4);
			a4 = ((float4*)&a_blk[l])[(threadIdx.x + ofs/4) & 0x1f];
			#pragma unroll
			for (int y = 0; y < 16; y += 4)
			{
				float4 dz4 = ((float4*)&dz_blk[l][(16*threadIdx.y + ofs + y) & 0x7f])[0];
				#pragma unroll
				for (int r = 0; r < 4; r++)
				{
					acc[y + r].x += ((float*)&dz4)[r]*a4.x;
					acc[y + r].y += ((float*)&dz4)[r]*a4.y;
					acc[y + r].z += ((float*)&dz4)[r]*a4.z;
					acc[y + r].w += ((float*)&dz4)[r]*a4.w;
				}
			}
			N--;
			if (N == 0)
				break;
		}
		__syncthreads();
	}
	
	int w_stride = W->stride;
	if (128*blockIdx.x + 4*threadIdx.x >= w_stride)
		return;
		
	float* w = W->data + (128*blockIdx.y + 16*threadIdx.y)*w_stride + 128*blockIdx.x + 4*threadIdx.x;
	float* w_end = W->data + W->rows*w_stride;
	int M = dZ->cols;
	for (int y = 0; y < 16; y++)
	{
		if (w >= w_end)
			return;
		float4 w4 = ((float4*)w)[0];
		w4.x -= K*acc[y].x/M;
		w4.y -= K*acc[y].y/M;
		w4.z -= K*acc[y].z/M;
		w4.w -= K*acc[y].w/M;
		((float4*)w)[0] = w4;
		w += w_stride;
	}
}